#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <vector>

#include <hipfft/hipfft.h>

#include "parameters.h"

using namespace std;

// Copied from docs
// HIPFFT_SUCCESS = 0, // The cuFFT operation was successful
// HIPFFT_INVALID_PLAN = 1, // cuFFT was passed an invalid plan handle
// HIPFFT_ALLOC_FAILED = 2, // cuFFT failed to allocate GPU or CPU memory
// HIPFFT_INVALID_TYPE = 3, // No longer used
// HIPFFT_INVALID_VALUE = 4, // User specified an invalid pointer or parameter
// HIPFFT_INTERNAL_ERROR = 5, // Driver or internal cuFFT library error
// HIPFFT_EXEC_FAILED = 6, // Failed to execute an FFT on the GPU
// HIPFFT_SETUP_FAILED = 7, // The cuFFT library failed to initialize
// HIPFFT_INVALID_SIZE = 8, // User specified an invalid transform size
// HIPFFT_UNALIGNED_DATA = 9, // No longer used
// HIPFFT_INCOMPLETE_PARAMETER_LIST = 10, // Missing parameters in call
// HIPFFT_INVALID_DEVICE = 11, // Execution of a plan was on different GPU than plan creation
// HIPFFT_PARSE_ERROR = 12, // Internal plan database error
// HIPFFT_NO_WORKSPACE = 13 // No workspace has been provided prior to plan execution
#define checkCufft(result) do {           \
    if (result != HIPFFT_SUCCESS) {                      \
        fprintf(stderr, "CUFFT at %d error: %d\n", __LINE__, result);   \
        exit(-1);                                       \
    }                                                   \
} while(0)

#define checkCuda(result) do {            \
    if (result != hipSuccess) {                        \
        fprintf(stderr, "CUDA at %d error: %d\n", __LINE__, result);   \
        exit(-1);                                       \
    }                                                   \
} while(0)

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

/*

computeGPU()
{

}

*/

vector<double> pricesAtPayoff(Parameters& prms)
{
    vector<double> out(prms.resolution);

    // Tree parameters (see p.53 of notes).
    double u = exp(prms.volatility * sqrt(prms.timeIncrement));
    double d = 1.0 / u;
    double a = exp(prms.riskFreeRate * prms.timeIncrement);
    // double p = (a - d) / (u - d);

    double N = prms.resolution;
    for (int i = 0; i < N; i++) {
        double asset = prms.startPrice * pow(u, i) * pow(d, N - i);
        if (prms.optionType == Call) {
            out[i] = max(asset - prms.strikePrice, 0.0);
        } else {
            out[i] = max(prms.strikePrice - asset, 0.0);
        }
    }

    return out;
}

// Prints Hello, World if the GPU code is working right.
void helloWorld()
{
    const int N = 16;
    const int blocksize = 16;

    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    hipMalloc( (void**)&ad, csize );
    hipMalloc( (void**)&bd, isize );
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );

    printf("%s\n", a);
}

void printPrices(vector<double>& prices) {
    for (int i = 0; i < prices.size(); i++) {
        printf("%f ", prices[i]);
    }
    printf("\n");
}

int main()
{
    assert(sizeof(hipfftDoubleReal) == sizeof(double));
    assert(sizeof(hipfftDoubleComplex) == 2 * sizeof(double));

    helloWorld();

    Parameters params;
    vector<double> prices = pricesAtPayoff(params);

    printPrices(prices);

    double N = params.resolution;

    hipfftDoubleReal* d_prices;
    checkCuda(hipMalloc((void**)&d_prices, sizeof(hipfftDoubleReal) * N));
    checkCuda(hipMemcpy(d_prices, &prices[0], sizeof(hipfftDoubleReal) * N,
                         hipMemcpyHostToDevice));

    hipfftDoubleComplex* d_ft;
    checkCuda(hipMalloc((void**)&d_ft, sizeof(hipfftDoubleComplex) * N));

    hipfftHandle plan;
    // Double to double-complex interleaved
    checkCufft(hipfftPlan1d(&plan, N, HIPFFT_D2Z, /* deprecated? */ 1));
    //checkCufft(hipfftPlan3d(&plan, 5, 5, 5, HIPFFT_C2C /* deprecated? */));

    // Forward transform
    checkCufft(hipfftExecD2Z(plan, d_prices, d_ft));

    // Reverse transform
    checkCufft(hipfftExecZ2D(plan, d_ft, d_prices));

    checkCuda(hipMemcpy(d_prices, &prices[0], sizeof(hipfftDoubleReal) * N,
                         hipMemcpyDeviceToHost));
    printPrices(prices);

    // Destroy the cuFFT plan.
    hipfftDestroy(plan);
    hipFree(d_prices);
    hipFree(d_ft);

    return EXIT_SUCCESS;
}

