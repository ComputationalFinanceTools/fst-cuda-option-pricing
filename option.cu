#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <vector>

#include <hipfft/hipfft.h>

#include "parameters.h"

using namespace std;

// Copied from docs
// HIPFFT_SUCCESS = 0, // The cuFFT operation was successful
// HIPFFT_INVALID_PLAN = 1, // cuFFT was passed an invalid plan handle
// HIPFFT_ALLOC_FAILED = 2, // cuFFT failed to allocate GPU or CPU memory
// HIPFFT_INVALID_TYPE = 3, // No longer used
// HIPFFT_INVALID_VALUE = 4, // User specified an invalid pointer or parameter
// HIPFFT_INTERNAL_ERROR = 5, // Driver or internal cuFFT library error
// HIPFFT_EXEC_FAILED = 6, // Failed to execute an FFT on the GPU
// HIPFFT_SETUP_FAILED = 7, // The cuFFT library failed to initialize
// HIPFFT_INVALID_SIZE = 8, // User specified an invalid transform size
// HIPFFT_UNALIGNED_DATA = 9, // No longer used
// HIPFFT_INCOMPLETE_PARAMETER_LIST = 10, // Missing parameters in call
// HIPFFT_INVALID_DEVICE = 11, // Execution of a plan was on different GPU than plan creation
// HIPFFT_PARSE_ERROR = 12, // Internal plan database error
// HIPFFT_NO_WORKSPACE = 13 // No workspace has been provided prior to plan execution
#define checkCufft(result) do {           \
    if (result != HIPFFT_SUCCESS) {                      \
        fprintf(stderr, "CUFFT at %d error: %d\n", __LINE__, result);   \
        exit(-1);                                       \
    }                                                   \
} while(0)

#define checkCuda(result) do {            \
    if (result != hipSuccess) {                        \
        fprintf(stderr, "CUDA at %d error: %d\n", __LINE__, result);   \
        exit(-1);                                       \
    }                                                   \
} while(0)

__host__ __device__ static __inline__
hipfftComplex cuComplexExponential(hipfftComplex x)
{
    float a = hipCrealf(x);
    float b = hipCrealf(x);
    float ea = exp(a);
    return make_hipComplex(ea * cos(b), ea * sin(b));
}

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

__global__
void solveODE(hipfftComplex* ft,
              float from_time,         // τ_l (T - t_l)
              float to_time,           // τ_u (T - t_u)
              float riskFreeRate, float volatility,
              float jumpMean, float kappa)
{
    int idx = threadIdx.x;

    hipfftComplex old_value = ft[idx];

    // Frequency.
    float k = 0.0;

    // Calculate Ψ (psi) (2.14)
    // Equation slightly simplified to save a few operations.
    float fst_term = volatility * M_PI * k;
    float psi_real = (-2.0 * fst_term * fst_term) - (riskFreeRate + jumpMean);
    float psi_imag = (riskFreeRate - jumpMean * kappa - volatility * volatility / 2.0) *
                      (2 * M_PI * k);

    // TODO: jump component.

    // Solution to ODE (2.27)
    float delta_tau = to_time - from_time;
    hipfftComplex exponent =
        make_hipComplex(psi_real * delta_tau, psi_imag * delta_tau);
    hipfftComplex exponential = cuComplexExponential(exponent);

    hipfftComplex new_value = hipCmulf(old_value, exponential);

    ft[idx] = new_value;
}

/*

computeGPU()
{

}

*/

vector<float> pricesAtPayoff(Parameters& prms)
{
    vector<float> out(prms.resolution);

    // Tree parameters (see p.53 of notes).
    float u = exp(prms.volatility * sqrt(prms.timeIncrement));
    float d = 1.0 / u;
    float a = exp(prms.riskFreeRate * prms.timeIncrement);
    // float p = (a - d) / (u - d);

    float N = prms.resolution;
    for (int i = 0; i < N; i++) {
        float asset = prms.startPrice * pow(u, i) * pow(d, N - i);
        if (prms.optionType == Call) {
            out[i] = max(asset - prms.strikePrice, 0.0);
        } else {
            out[i] = max(prms.strikePrice - asset, 0.0);
        }
    }

    return out;
}

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %zu\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %zu\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %zu\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %zu\n", devProp.totalConstMem);
    printf("Texture alignment:             %zu\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

void printAllDevices()
{
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
}

// Prints Hello, World if the GPU code is working right.
void helloWorld()
{
    const int N = 16;
    const int blocksize = 16;

    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    checkCuda(hipMalloc( (void**)&ad, csize ));
    checkCuda(hipMalloc( (void**)&bd, isize ));
    checkCuda(hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ));
    checkCuda(hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ));

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    checkCuda(hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ));
    checkCuda(hipFree( ad ));
    checkCuda(hipFree( bd ));

    printf("%s\n", a);
}

// Run a couple tests to see that CUDA works properly.
void cudaCheck()
{
    printf("Calling hipFree(0) no-op...\n");
    hipFree(0);
    printf("Calling hipFree(0) succeeded!\n");

    printAllDevices();
    helloWorld();
}

void printPrices(vector<float>& prices) {
    for (int i = 0; i < prices.size(); i++) {
        printf("%f ", prices[i]);
    }
    printf("\n");
}

int main()
{
    assert(sizeof(hipfftReal) == sizeof(float));
    assert(sizeof(hipfftComplex) == 2 * sizeof(float));

    cudaCheck();

    printf("\nChecks finished. Starting option calculation...\n\n");

    Parameters params;
    vector<float> prices = pricesAtPayoff(params);

    printPrices(prices);

    float N = params.resolution;

    hipfftReal* d_prices;
    checkCuda(hipMalloc((void**)&d_prices, sizeof(hipfftReal) * N));
    checkCuda(hipMemcpy(d_prices, &prices[0], sizeof(hipfftReal) * N,
                         hipMemcpyHostToDevice));

    hipfftComplex* d_ft;
    checkCuda(hipMalloc((void**)&d_ft, sizeof(hipfftComplex) * N));

    hipfftHandle plan;
    hipfftHandle planr;

    // Float to complex interleaved
    checkCufft(hipfftPlan1d(&plan, N, HIPFFT_R2C, /* deprecated? */ 1));
    checkCufft(hipfftPlan1d(&planr, N, HIPFFT_C2R, /* deprecated? */ 1));

    // Forward transform
    checkCufft(hipfftExecR2C(plan, d_prices, d_ft));

    // Solve ODE
    solveODE<<<dim3(N, 1), dim3(1, 1)>>>(d_ft, 0.0, params.expiryTime,
            params.riskFreeRate,
            params.volatility, params.jumpMean, params.kappa);

    // Reverse transform
    checkCufft(hipfftExecC2R(planr, d_ft, d_prices));

    checkCuda(hipMemcpy(d_prices, &prices[0], sizeof(hipfftReal) * N,
                         hipMemcpyDeviceToHost));
    printPrices(prices);

    // Destroy the cuFFT plan.
    hipfftDestroy(plan);
    hipFree(d_prices);
    hipFree(d_ft);

    return EXIT_SUCCESS;
}

