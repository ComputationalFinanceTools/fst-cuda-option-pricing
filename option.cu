#include "hip/hip_runtime.h"
#include <assert.h>
#include <getopt.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <vector>

#include <hipfft/hipfft.h>

#include "parameters.h"

using namespace std;

// Copied from docs
// HIPFFT_SUCCESS = 0, // The cuFFT operation was successful
// HIPFFT_INVALID_PLAN = 1, // cuFFT was passed an invalid plan handle
// HIPFFT_ALLOC_FAILED = 2, // cuFFT failed to allocate GPU or CPU memory
// HIPFFT_INVALID_TYPE = 3, // No longer used
// HIPFFT_INVALID_VALUE = 4, // User specified an invalid pointer or parameter
// HIPFFT_INTERNAL_ERROR = 5, // Driver or internal cuFFT library error
// HIPFFT_EXEC_FAILED = 6, // Failed to execute an FFT on the GPU
// HIPFFT_SETUP_FAILED = 7, // The cuFFT library failed to initialize
// HIPFFT_INVALID_SIZE = 8, // User specified an invalid transform size
// HIPFFT_UNALIGNED_DATA = 9, // No longer used
// HIPFFT_INCOMPLETE_PARAMETER_LIST = 10, // Missing parameters in call
// HIPFFT_INVALID_DEVICE = 11, // Execution of a plan was on different GPU than plan creation
// HIPFFT_PARSE_ERROR = 12, // Internal plan database error
// HIPFFT_NO_WORKSPACE = 13 // No workspace has been provided prior to plan execution
#define checkCufft(result) do {           \
    if (result != HIPFFT_SUCCESS) {                      \
        fprintf(stderr, "CUFFT at %d error: %d\n", __LINE__, result);   \
        exit(-1);                                       \
    }                                                   \
} while(0)

#define checkCuda(result) do {            \
    if (result != hipSuccess) {                        \
        fprintf(stderr, "CUDA at %d error: %d\n", __LINE__, result);   \
        exit(-1);                                       \
    }                                                   \
} while(0)

__host__ __device__ static __inline__
hipfftComplex cuComplexExponential(hipfftComplex x)
{
    float a = hipCrealf(x);
    float b = hipCimagf(x);
    float ea = exp(a);
    return make_hipComplex(ea * cos(b), ea * sin(b));
}

__host__ __device__ static __inline__
hipfftComplex cuComplexScalarMult(float scalar, hipfftComplex x)
{
    float a = hipCrealf(x);
    float b = hipCimagf(x);
    return make_hipComplex(scalar * a, scalar * b);
}

__global__
void hello(char *a, int *b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] += b[idx];
}

__global__
void normalize(hipfftReal* ft, int length)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    ft[idx] /= length;
}

__global__
void solveODE(hipfftComplex* ft,
              float from_time,         // τ_l (T - t_l)
              float to_time,           // τ_u (T - t_u)
              float riskFreeRate, float volatility,
              float jumpMean, float kappa,
              int N, float delta_frequency)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hipfftComplex old_value = ft[idx];

    // Frequency (see p.11 for discretization).
    float m;
    if (idx <= N / 2) {
        m = idx;
    } else {
        m = idx - N;
    }
    float k = delta_frequency * m;

    // Calculate Ψ (psi) (2.14)
    // Equation slightly simplified to save a few operations.
    float fst_term = volatility * M_PI * k;
    float psi_real = (-2.0 * fst_term * fst_term) - (riskFreeRate + jumpMean);
    float psi_imag = (riskFreeRate - jumpMean * kappa - volatility * volatility / 2.0) *
                      (2 * M_PI * k);

    // TODO: jump component.

    // Solution to ODE (2.27)
    float delta_tau = to_time - from_time;
    hipfftComplex exponent =
        make_hipComplex(psi_real * delta_tau, psi_imag * delta_tau);
    hipfftComplex exponential = cuComplexExponential(exponent);

    hipfftComplex new_value = hipCmulf(old_value, exponential);

    ft[idx] = new_value;
}

vector<float> assetPricesAtPayoff(Parameters& prms)
{
    float N = prms.resolution;
    vector<float> out(N);

    float x_max = prms.logBoundary;
    float x_min = -prms.logBoundary;
    float delta_x = (x_max - x_min) / (N - 1);

    /*
    // Tree parameters (see p.53 of notes).
    float u = exp(prms.volatility * sqrt(prms.timeIncrement));
    float d = 1.0 / u;
    float a = exp(prms.riskFreeRate * prms.timeIncrement);
    // float p = (a - d) / (u - d);

    for (int i = 0; i < N; i++) {
        out[i] = prms.startPrice * pow(u, i) * pow(d, N - i);
    }
    */

    for (int i = 0; i < N; i++) {
        out[i] = prms.startPrice * exp(x_min + i * delta_x);
    }

    return out;
}

vector<float> optionValuesAtPayoff(Parameters& prms, vector<float>& assetPrices)
{
    vector<float> out(prms.resolution);

    float N = prms.resolution;
    for (int i = 0; i < N; i++) {
        if (prms.optionPayoffType == Call) {
            out[i] = max(assetPrices[i] - prms.strikePrice, 0.0);
        } else {
            out[i] = max(prms.strikePrice - assetPrices[i], 0.0);
        }
    }

    return out;
}

void printComplex(hipfftComplex x) {
    float a = hipCrealf(x);
    float b = hipCimagf(x);
    printf("%f + %fi", a, b);
}

void printComplexArray(vector<hipfftComplex> xs)
{
    for (int i = 0; i < xs.size(); i++) {
        printComplex(xs[i]);
        if (i < xs.size() - 1)
            printf(", ");
        if (i % 5 == 0 && i > 0)
            printf("\n");
    }
    printf("\n");
}

vector<hipfftComplex> dft(vector<float>& in)
{
    vector<hipfftComplex> out(in.size());

    for (int k = 0; k < out.size(); k++) {
        out[k] = make_hipComplex(0, 0);

        for (int n = 0; n < in.size(); n++) {
            hipfftComplex exponent = make_hipComplex(0, -2.0f * M_PI * k * n / in.size());
            out[k] = hipCaddf(out[k], cuComplexScalarMult(in[n], cuComplexExponential(exponent)));
        }
    }

    return out;
}

vector<hipfftComplex> idft_complex(vector<hipfftComplex>& in)
{
    vector<hipfftComplex> out(in.size());

    for (int k = 0; k < out.size(); k++) {
        out[k] = make_hipComplex(0, 0);

        for (int n = 0; n < in.size(); n++) {
            hipfftComplex exponent = make_hipComplex(0, 2.0f * M_PI * k * n / in.size());
            out[k] = hipCaddf(out[k], hipCmulf(in[n], cuComplexExponential(exponent)));
        }

        out[k] = cuComplexScalarMult(1.0 / out.size(), out[k]);
    }

    /*
    printComplexArray(out);
    printf("\n");
    */

    return out;
}

vector<float> idft(vector<hipfftComplex>& in)
{
    vector<hipfftComplex> ift = idft_complex(in);
    vector<float> out(ift.size());
    for (int i = 0; i < ift.size(); i++) {
        out[i] = hipCrealf(ift[i]);
    }
    return out;
}

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %zu\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %zu\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %zu\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %zu\n", devProp.totalConstMem);
    printf("Texture alignment:             %zu\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

void printAllDevices()
{
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // For some reason, systems that don't have CUDA devices might
    // print infinitely many of them. If we run the program accidently,
    // the program might hang while printing. We don't want that.
    if (devCount > 5)
        printf("Printing first 5 devices.\n");

    // Iterate through devices
    for (int i = 0; i < min(5, devCount); ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
}

// Prints Hello, World if the GPU code is working right.
void helloWorld()
{
    const int N = 16;
    const int blocksize = 16;

    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    checkCuda(hipMalloc( (void**)&ad, csize ));
    checkCuda(hipMalloc( (void**)&bd, isize ));
    checkCuda(hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ));
    checkCuda(hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ));

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    checkCuda(hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ));
    checkCuda(hipFree( ad ));
    checkCuda(hipFree( bd ));

    printf("%s\n", a);
}

// Run a couple tests to see that CUDA works properly.
void cudaCheck()
{
    printf("Calling hipFree(0) no-op...\n");
    hipFree(0);
    printf("Calling hipFree(0) succeeded!\n");

    printAllDevices();
    helloWorld();
}

void printPrices(vector<float>& prices) {
    for (int i = 0; i < prices.size(); i++) {
        printf("%f ", prices[i]);
    }
    printf("\n");
}

void computeCPU(Parameters& params, vector<float>& assetPrices, vector<float>& optionValues)
{
    int N = params.resolution;

    // Discretization parameters (see p.11)
    float x_max = params.logBoundary;
    float x_min = -params.logBoundary;
    float delta_frequency = (float)(N - 1) / (x_max - x_min) / N;

    float from_time = 0.0f;
    float to_time = params.expiryTime;
    float riskFreeRate = params.riskFreeRate;
    float volatility = params.volatility;
    float jumpMean = params.jumpMean;
    float kappa = params.kappa;

    // Forward transform
    vector<hipfftComplex> ft = dft(optionValues);
    vector<hipfftComplex> ft2(N);

    for (int idx = 0; idx < ft.size(); idx++) {
        hipfftComplex old_value = ft[idx];

        // Frequency (see p.11 for discretization).
        float m;
        if (idx <= N / 2) {
            m = idx;
        } else {
            m = idx - N;
        }
        float k = delta_frequency * m;

        // Calculate Ψ (psi) (2.14)
        // Equation slightly simplified to save a few operations.
        float fst_term = volatility * M_PI * k;
        float psi_real = (-2.0 * fst_term * fst_term) - (riskFreeRate + jumpMean);
        float psi_imag = (riskFreeRate - jumpMean * kappa - volatility * volatility / 2.0) *
                          (2 * M_PI * k);

        // TODO: jump component.

        // Solution to ODE (2.27)
        float delta_tau = to_time - from_time;
        hipfftComplex exponent =
            make_hipComplex(psi_real * delta_tau, psi_imag * delta_tau);
        hipfftComplex exponential = cuComplexExponential(exponent);

        hipfftComplex new_value = hipCmulf(old_value, exponential);

        ft2[idx] = new_value;
    }

    // Inverse transform
    vector<float> ift = idft(ft2);

    // printPrices(ift);

    float answer_index = -x_min * (N - 1) / (x_max - x_min);
    float price_lower = ift[(int)floor(answer_index)];
    float price_upper = ift[(int)ceil(answer_index)];
    float interpolated = price_lower * (ceil(answer_index) - answer_index) +
                         price_upper * (answer_index - floor(answer_index));
    printf("Price is at index %f. Price at %d: %f. Price at %d: %f.\n",
            answer_index, (int)floor(answer_index), price_lower,
            (int)ceil(answer_index), price_upper);
    printf("Interpolated price: %f\n", interpolated);
}

void computeGPU(Parameters& params, vector<float>& assetPrices, vector<float>& optionValues)
{
    // Option values at time t = 0
    vector<float> initialValues(optionValues.size());

    float N = params.resolution;

    hipfftReal* d_prices;
    checkCuda(hipMalloc((void**)&d_prices, sizeof(hipfftReal) * N));
    checkCuda(hipMemcpy(d_prices, &optionValues[0], sizeof(hipfftReal) * N,
                         hipMemcpyHostToDevice));

    hipfftComplex* d_ft;
    checkCuda(hipMalloc((void**)&d_ft, sizeof(hipfftComplex) * N));

    hipfftHandle plan;
    hipfftHandle planr;

    // Float to complex interleaved
    checkCufft(hipfftPlan1d(&plan, N, HIPFFT_R2C, /* deprecated? */ 1));
    checkCufft(hipfftPlan1d(&planr, N, HIPFFT_C2R, /* deprecated? */ 1));

    // Discretization parameters (see p.11)
    float x_max = params.logBoundary;
    float x_min = -params.logBoundary;
    float delta_frequency = (float)(N - 1) / (x_max - x_min) / N;

    for (int i = 0; i < params.timesteps; i++) {
        float from_time = (float)i / params.timesteps * params.expiryTime;
        float to_time = (float)(i + 1) / params.timesteps * params.expiryTime;

        // Forward transform
        checkCufft(hipfftExecR2C(plan, d_prices, d_ft));

        // Solve ODE
        // Note that we solve the ODE only on the first half of the frequency
        // data. Why? A fourier transform on real (non-complex) data will give
        // hermetian symmetry, where the second half of the array is just the
        // complex conjugate of the first half. So cufft & fftw doesn't store
        // any values in the second half at all! They don't use the second half
        // of the array either to compute the inverse fourier transform.
        // See http://www.fftw.org/doc/The-1d-Real_002ddata-DFT.html
        int ode_size = N / 2;
        solveODE<<<dim3(ode_size / 512, 1), dim3(min(ode_size, 512), 1)>>>(
                d_ft, from_time, to_time, params.riskFreeRate,
                params.volatility, params.jumpMean, params.kappa,
                N, delta_frequency);

        // Reverse transform
        checkCufft(hipfftExecC2R(planr, d_ft, d_prices));
        normalize<<<dim3(N / 512, 1), dim3(min((int)N, 512), 1)>>>(d_prices, N);
    }

    checkCuda(hipMemcpy(&initialValues[0], d_prices, sizeof(hipfftReal) * N,
                         hipMemcpyDeviceToHost));

    // Destroy the cuFFT plan.
    hipfftDestroy(plan);
    hipfftDestroy(planr);
    hipFree(d_prices);
    hipFree(d_ft);

    float answer_index = -x_min * (N - 1) / (x_max - x_min);
    float price_lower = initialValues[(int)floor(answer_index)];
    float price_upper = initialValues[(int)ceil(answer_index)];
    float interpolated = price_lower * (ceil(answer_index) - answer_index) +
                         price_upper * (answer_index - floor(answer_index));
    printf("Price is at index %f. Price at %d: %f. Price at %d: %f.\n",
            answer_index, (int)floor(answer_index), price_lower,
            (int)ceil(answer_index), price_upper);
    printf("Interpolated price: %f\n", interpolated);
}

int main(int argc, char** argv)
{
    assert(sizeof(hipfftReal) == sizeof(float));
    assert(sizeof(hipfftComplex) == 2 * sizeof(float));

    Parameters params;

    // Parse arguments
    while (true) {
        static struct option long_options[] = {
            {"payoff",  required_argument, 0, 'p'},
            {"exercise",  required_argument, 0, 'e'},
            {"timesteps",  required_argument, 0, 't'},
            {0, 0, 0, 0}
        };

        int option_index = 0;
        char c = getopt_long(argc, argv, "abc:d:f:", long_options, &option_index);

        if (c == -1) {
            break;
        }

        switch (c) {
            case 'e':
                if (!strcmp(optarg, "european")) {
                    params.optionExerciseType = European;
                } else if (!strcmp(optarg, "american")) {
                    params.optionExerciseType = American;
                } else {
                    fprintf(stderr, "Option exercise type %s invalid.\n", optarg);
                    abort();
                }
            case 'p':
                if (!strcmp(optarg, "put")) {
                    params.optionPayoffType = Put;
                } else if (!strcmp(optarg, "call")) {
                    params.optionPayoffType = Call;
                } else {
                    fprintf(stderr, "Option payoff type %s invalid.\n", optarg);
                    abort();
                }
            case 't':
                params.timesteps = atoi(optarg);
            case '?':
                break;
            default:
                abort();
        }
    }

    cudaCheck();

    printf("\nChecks finished. Starting option calculation...\n\n");

    vector<float> assetPrices = assetPricesAtPayoff(params);
    vector<float> optionValues = optionValuesAtPayoff(params, assetPrices);

    printPrices(optionValues);

    printf("\nComputing CPU results...\n");
    //computeCPU(params, assetPrices, optionValues);
    printf("\nComputing GPU results...\n");
    computeGPU(params, assetPrices, optionValues);

    /*
    float X[] = {1, 0, 1, 0, 1, 0, 1, 0};
    vector<float> a(X, X + 8);
    vector<hipfftComplex> x = dft(a);
    printComplexArray(x);
    vector<hipfftComplex> y = idft(x);
    printComplexArray(y);
    */

    return EXIT_SUCCESS;
}

