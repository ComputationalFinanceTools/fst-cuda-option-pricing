#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hipfft/hipfft.h>

#include "parameters.h"

const int N = 16;
const int blocksize = 16;

#define NX 64
#define NY 64
#define NZ 128

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

/*

computeGPU()
{

}

*/

int main()
{
    Parameters params;

    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    hipMalloc( (void**)&ad, csize );
    hipMalloc( (void**)&bd, isize );
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );

    printf("%s\n", a);

    hipfftHandle plan;
    hipfftComplex *data1, *data2;
    hipMalloc((void**)&data1, sizeof(hipfftComplex)*NX*NY*NZ);
    hipMalloc((void**)&data2, sizeof(hipfftComplex)*NX*NY*NZ);
    // Create a 3D FFT plan.
    hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

    // Transform the first signal in place.
    hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

    // Transform the second signal using the same plan.
    hipfftExecC2C(plan, data2, data2, HIPFFT_FORWARD);

    // Destroy the cuFFT plan.
    hipfftDestroy(plan);
    hipFree(data1); hipFree(data2);

    return EXIT_SUCCESS;
}

