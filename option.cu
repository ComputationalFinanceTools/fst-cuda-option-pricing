#include "hip/hip_runtime.h"
#include <assert.h>
#include <getopt.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <vector>

#include <hipfft/hipfft.h>

#include "parameters.h"
#include "utils.h"

#ifdef USE_FLOAT

// For quick testing of doubles only, otherwise this is obviously a terrible idea.
#define double float
#define complex hipfftComplex
#define hipCreal hipCrealf
#define hipCimag hipCimagf
#define hipCadd hipCaddf
#define hipCmul hipCmulf
#define hipCdiv hipCdivf
#define hipConj hipConjf
#define HIPFFT_D2Z HIPFFT_R2C
#define HIPFFT_Z2D HIPFFT_C2R
#define hipfftExecD2Z hipfftExecR2C
#define hipfftExecZ2D hipfftExecC2R
#define makeComplex make_hipComplex

#else

#define complex hipfftDoubleComplex
#define makeComplex make_hipDoubleComplex

#endif

using namespace std;

__host__ __device__ static __inline__
complex cuComplexExponential(complex x)
{
    double a = hipCreal(x);
    double b = hipCimag(x);
    double ea = exp(a);
    return makeComplex(ea * cos(b), ea * sin(b));
}

__host__ __device__ static __inline__
complex cuComplexScalarMult(double scalar, complex x)
{
    double a = hipCreal(x);
    double b = hipCimag(x);
    return makeComplex(scalar * a, scalar * b);
}

__global__
void normalize(double* ft, int length)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    ft[idx] /= length;
}

__global__
// TODO: Need better argument names for the last two...
void earlyExercise(double* ft, double startPrice, double strikePrice,
                   double x_min, double delta_x,
                   OptionPayoffType type)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double assetPrice = startPrice * exp(x_min + idx * delta_x);
    if (type == Call) {
        ft[idx] = max(ft[idx], max(assetPrice - strikePrice, 0.0));
    } else {
        ft[idx] = max(ft[idx], max(strikePrice - assetPrice, 0.0));
    }
}

__global__
void solveODE(complex* ft,
              complex* jump_ft,   // Fourier transform of the jump function
              double from_time,         // τ_l (T - t_l)
              double to_time,           // τ_u (T - t_u)
              double riskFreeRate,
              double dividend,
              double volatility,
              double jumpMean,
              double kappa,
              double delta_frequency,
              int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    complex old_value = ft[idx];

    // Frequency (see p.11 for discretization).
    double m;
    if (idx <= N / 2) {
        m = idx;
    } else {
        m = idx - N;
    }
    double k = delta_frequency * m;

    // Calculate Ψ (psi) (2.14)
    // The dividend is shown on p.13
    // Equation slightly simplified to save a few operations.
    // TODO: Continuous dividends?
    double fst_term = volatility * M_PI * k;
    complex psi = makeComplex(
            (-2.0 * fst_term * fst_term) - (riskFreeRate + jumpMean),
            (riskFreeRate - dividend - jumpMean * kappa - volatility * volatility / 2.0) *
                      (2 * M_PI * k));

    // Jump component.
    if (jump_ft) {
        psi = hipCadd(psi, cuComplexScalarMult(jumpMean, hipConj(jump_ft[idx])));
    }

    // Solution to ODE (2.27)
    double delta_tau = to_time - from_time;
    complex exponent = cuComplexScalarMult(delta_tau, psi);
    complex exponential = cuComplexExponential(exponent);

    complex new_value = hipCmul(old_value, exponential);

    ft[idx] = new_value;
}

vector<double> assetPricesAtPayoff(Parameters& prms)
{
    double N = prms.resolution;
    vector<double> out(N);

    // Discretization parameters (see p.11)
    // TODO: Factor out into params?
    double x_max = prms.logBoundary;
    double x_min = -prms.logBoundary;
    double delta_x = (x_max - x_min) / (N - 1);

    /*
    // Tree parameters (see p.53 of notes).
    double u = exp(prms.volatility * sqrt(prms.timeIncrement));
    double d = 1.0 / u;
    double a = exp(prms.riskFreeRate * prms.timeIncrement);
    // double p = (a - d) / (u - d);

    for (int i = 0; i < N; i++) {
        out[i] = prms.startPrice * pow(u, i) * pow(d, N - i);
    }
    */

    for (int i = 0; i < N; i++) {
        out[i] = prms.startPrice * exp(x_min + i * delta_x);
    }

    return out;
}

vector<double> optionValuesAtPayoff(Parameters& prms, vector<double>& assetPrices)
{
    vector<double> out(prms.resolution);

    double N = prms.resolution;
    for (int i = 0; i < N; i++) {
        if (prms.optionPayoffType == Call) {
            out[i] = max(assetPrices[i] - prms.strikePrice, 0.0);
        } else {
            out[i] = max(prms.strikePrice - assetPrices[i], 0.0);
        }
    }

    return out;
}

// Fourier transform of the Merton jump function.
vector<complex> mertonJumpFT(Parameters& prms, double delta_frequency)
{
    int N = prms.resolution;

    // See Lippa (2013) p.13
    vector<complex> ft(N);
    for (int i = 0; i < N; i++) {
        // Frequency (see p.11 for discretization).
        double m;
        if (i <= N / 2) {
            m = i;
        } else {
            m = i - N;
        }
        double k = delta_frequency * m;

        double real = M_PI * k * prms.mertonNormalStdev;
        real = -2 * real * real;
        double imag = -2 * M_PI * k * prms.driftRate;
        complex exponent = makeComplex(real, imag);
        ft[i] = cuComplexExponential(exponent);
    }

    return ft;
}

// Fourier transform of the Kou jump function
vector<complex> kouJumpFT(Parameters& prms, double delta_frequency)
{
    int N = prms.resolution;
    double p = prms.kouUpJumpProbability;

    // See Lippa (2013) p.54
    vector<complex> ft(N);
    for (int i = 0; i < N; i++) {
        // Frequency (see p.11 for discretization).
        double m;
        if (i <= N / 2) {
            m = i;
        } else {
            m = i - N;
        }
        double k = delta_frequency * m;

        complex up = hipCdiv(makeComplex(p, 0),
                makeComplex(1, 2 * M_PI * k / prms.kouUpRate));
        complex down = hipCdiv(makeComplex(1 - p, 0),
                makeComplex(1, -2 * M_PI * k / prms.kouDownRate));

        ft[i] = hipCadd(up, down);
    }

    return ft;
}

void printComplex(complex x) {
    double a = hipCreal(x);
    double b = hipCimag(x);
    printf("%f + %fi", a, b);
}

void printComplexArray(vector<complex> xs)
{
    for (int i = 0; i < xs.size(); i++) {
        printComplex(xs[i]);
        if (i < xs.size() - 1)
            printf(", ");
        if (i % 5 == 0 && i > 0)
            printf("\n");
    }
    printf("\n");
}

vector<complex> dft(vector<double>& in)
{
    vector<complex> out(in.size());

    for (int k = 0; k < out.size(); k++) {
        out[k] = makeComplex(0, 0);

        for (int n = 0; n < in.size(); n++) {
            complex exponent = makeComplex(0, -2.0f * M_PI * k * n / in.size());
            out[k] = hipCadd(out[k], cuComplexScalarMult(in[n], cuComplexExponential(exponent)));
        }
    }

    return out;
}

vector<complex> idft_complex(vector<complex>& in)
{
    vector<complex> out(in.size());

    for (int k = 0; k < out.size(); k++) {
        out[k] = makeComplex(0, 0);

        for (int n = 0; n < in.size(); n++) {
            complex exponent = makeComplex(0, 2.0f * M_PI * k * n / in.size());
            out[k] = hipCadd(out[k], hipCmul(in[n], cuComplexExponential(exponent)));
        }

        out[k] = cuComplexScalarMult(1.0 / out.size(), out[k]);
    }

    /*
    printComplexArray(out);
    printf("\n");
    */

    return out;
}

vector<double> idft(vector<complex>& in)
{
    vector<complex> ift = idft_complex(in);
    vector<double> out(ift.size());
    for (int i = 0; i < ift.size(); i++) {
        out[i] = hipCreal(ift[i]);
    }
    return out;
}

void printPrices(vector<double>& prices) {
    int first_negative = -1;
    for (int i = 0; i < prices.size(); i++) {
        printf("%f ", prices[i]);
        if (first_negative == -1 && prices[i] < 0) {
            first_negative = i;
        }
    }
    printf("\n");
    printf("First negative number at %d.\n", first_negative);
}

void computeCPU(Parameters& params, vector<double>& assetPrices, vector<double>& optionValues)
{
    int N = params.resolution;

    // Discretization parameters (see p.11)
    double x_max = params.logBoundary;
    double x_min = -params.logBoundary;
    double delta_frequency = (double)(N - 1) / (x_max - x_min) / N;

    double from_time = 0.0f;
    double to_time = params.expiryTime;
    double riskFreeRate = params.riskFreeRate;
    double volatility = params.volatility;
    double jumpMean = params.jumpMean;
    double kappa = params.kappa();

    // Forward transform
    vector<complex> ft = dft(optionValues);
    vector<complex> ft2(N);

    for (int idx = 0; idx < ft.size(); idx++) {
        complex old_value = ft[idx];

        // Frequency (see p.11 for discretization).
        double m;
        if (idx <= N / 2) {
            m = idx;
        } else {
            m = idx - N;
        }
        double k = delta_frequency * m;

        // Calculate Ψ (psi) (2.14)
        // Equation slightly simplified to save a few operations.
        double fst_term = volatility * M_PI * k;
        double psi_real = (-2.0 * fst_term * fst_term) - (riskFreeRate + jumpMean);
        double psi_imag = (riskFreeRate - jumpMean * kappa - volatility * volatility / 2.0) *
                          (2 * M_PI * k);

        // TODO: jump component.

        // Solution to ODE (2.27)
        double delta_tau = to_time - from_time;
        complex exponent =
            makeComplex(psi_real * delta_tau, psi_imag * delta_tau);
        complex exponential = cuComplexExponential(exponent);

        complex new_value = hipCmul(old_value, exponential);

        ft2[idx] = new_value;
    }

    // Inverse transform
    vector<double> ift = idft(ft2);

    // printPrices(ift);

    double answer_index = -x_min * (N - 1) / (x_max - x_min);
    double price_lower = ift[(int)floor(answer_index)];
    double price_upper = ift[(int)ceil(answer_index)];
    double interpolated = price_lower * (ceil(answer_index) - answer_index) +
                         price_upper * (answer_index - floor(answer_index));

    if (params.verbose) {
        printf("Price is at index %f. Price at %d: %f. Price at %d: %f.\n",
                answer_index, (int)floor(answer_index), price_lower,
                (int)ceil(answer_index), price_upper);
        printf("Interpolated price: %f\n", interpolated);
    } else {
        printf("%f\n", interpolated);
    }
}

void computeGPU(Parameters& params, vector<double>& assetPrices, vector<double>& optionValues)
{
    // Option values at time t = 0
    vector<double> initialValues(optionValues.size());

    int N = params.resolution;

    double* d_prices;
    checkCuda(hipMalloc((void**)&d_prices, sizeof(double) * N));
    checkCuda(hipMemcpy(d_prices, &optionValues[0], sizeof(double) * N,
                         hipMemcpyHostToDevice));

    complex* d_ft;
    checkCuda(hipMalloc((void**)&d_ft, sizeof(complex) * N));

    hipfftHandle plan;
    hipfftHandle planr;

    // Float to complex interleaved
    checkCufft(hipfftPlan1d(&plan, N, HIPFFT_D2Z, /* deprecated? */ 1));
    checkCufft(hipfftPlan1d(&planr, N, HIPFFT_Z2D, /* deprecated? */ 1));

    // Discretization parameters (see p.11)
    double x_max = params.logBoundary;
    double x_min = -params.logBoundary;
    double delta_x = (x_max - x_min) / (N - 1);
    double delta_frequency = (double)(N - 1) / (x_max - x_min) / N;

    // Jump function
    vector<complex> jump_ft;
    complex *d_jump_ft = NULL;

    if (params.jumpType == Merton) {
        jump_ft = mertonJumpFT(params, delta_frequency);
    } else if (params.jumpType == Kou) {
        jump_ft = kouJumpFT(params, delta_frequency);
    }

    if (params.jumpType != None) {
        checkCuda(hipMalloc((void**)&d_jump_ft, sizeof(complex) * N));
        checkCuda(hipMemcpy(d_jump_ft, &jump_ft[0], sizeof(complex) * N,
                             hipMemcpyHostToDevice));
    }

    for (int i = 0; i < params.timesteps; i++) {
        double from_time = (double)i / params.timesteps * params.expiryTime;
        double to_time = (double)(i + 1) / params.timesteps * params.expiryTime;

        // Forward transform
        checkCufft(hipfftExecD2Z(plan, d_prices, d_ft));

        // Solve ODE
        // Note that we solve the ODE only on the first half of the frequency
        // data. Why? A fourier transform on real (non-complex) data will give
        // hermetian symmetry, where the second half of the array is just the
        // complex conjugate of the first half. So cufft & fftw doesn't store
        // any values in the second half at all! They don't use the second half
        // of the array either to compute the inverse fourier transform.
        // See http://www.fftw.org/doc/The-1d-Real_002ddata-DFT.html
        int ode_size = N / 2 + 1;
        solveODE<<<dim3(max(ode_size / 512, 1), 1), dim3(min(ode_size, 512), 1)>>>(
                d_ft, d_jump_ft, from_time, to_time,
                params.riskFreeRate, params.dividendRate,
                params.volatility, params.jumpMean, params.kappa(),
                delta_frequency, N);

        // Reverse transform
        checkCufft(hipfftExecZ2D(planr, d_ft, d_prices));
        normalize<<<dim3(max(N / 512, 1), 1), dim3(min((int)N, 512), 1)>>>(d_prices, N);

        // Consider early exercise for American options. This is the same technique
        // as option pricing using dynamic programming: at each timestep, set the
        // option value to the payoff if is higher than the current option value.
        if (params.optionExerciseType == American) {
            earlyExercise<<<dim3(max(N / 512, 1), 1), dim3(min((int)N, 512), 1)>>>(
                    d_prices, params.startPrice, params.strikePrice,
                    x_min, delta_x, params.optionPayoffType);
        }
    }

    checkCuda(hipMemcpy(&initialValues[0], d_prices, sizeof(double) * N,
                         hipMemcpyDeviceToHost));

    // Destroy the cuFFT plan.
    hipfftDestroy(plan);
    hipfftDestroy(planr);
    hipFree(d_prices);
    hipFree(d_ft);
    hipFree(d_jump_ft);

    double answer_index = -x_min * (N - 1) / (x_max - x_min);
    double price_lower = initialValues[(int)floor(answer_index)];
    double price_upper = initialValues[(int)ceil(answer_index)];
    double interpolated = price_lower * (ceil(answer_index) - answer_index) +
                         price_upper * (answer_index - floor(answer_index));

    if (params.verbose) {
        printf("Price is at index %f. Price at %d: %f. Price at %d: %f.\n",
                answer_index, (int)floor(answer_index), price_lower,
                (int)ceil(answer_index), price_upper);
        printf("Interpolated price: %f\n", interpolated);
    } else {
        printf("%f\n", interpolated);
    }
}

int main(int argc, char** argv)
{
    assert(sizeof(complex) == 2 * sizeof(double));

    Parameters params;

    // Parse arguments
    while (true) {
        static struct option long_options[] = {
            {"payoff",  required_argument, 0, 'p'},
            {"exercise",  required_argument, 0, 'e'},
            {"dividend",  required_argument, 0, 'q'},
            {"debug",  no_argument, 0, 'd'},
            {"mertonjumps",  no_argument, 0, 'm'},
            {"koujumps",  no_argument, 0, 'k'},
            {"lambda",  required_argument, 0, 'l'},
            {"p",  required_argument, 0, '0'},
            {"eta1",  required_argument, 0, '1'},
            {"eta2",  required_argument, 0, '2'},
            {"gamma",  required_argument, 0, 'y'},
            {"S",  required_argument, 0, 'S'},
            {"K",  required_argument, 0, 'K'},
            {"r",  required_argument, 0, 'r'},
            {"T",  required_argument, 0, 'T'},
            {"sigma",  required_argument, 0, 'o'},
            {"resolution",  required_argument, 0, 'n'},
            {"timesteps",  required_argument, 0, 't'},
            {"verbose",  no_argument, 0, 'v'},
            {0, 0, 0, 0}
        };

        int option_index = 0;
        char c = getopt_long(argc, argv, "abc:d:f:", long_options, &option_index);

        if (c == -1) {
            break;
        }

        switch (c) {
            case 'e':
                if (!strcmp(optarg, "european")) {
                    params.optionExerciseType = European;
                } else if (!strcmp(optarg, "american")) {
                    params.optionExerciseType = American;
                } else {
                    fprintf(stderr, "Option exercise type %s invalid.\n", optarg);
                    abort();
                }
                break;
            case 'p':
                if (!strcmp(optarg, "put")) {
                    params.optionPayoffType = Put;
                } else if (!strcmp(optarg, "call")) {
                    params.optionPayoffType = Call;
                } else {
                    fprintf(stderr, "Option payoff type %s invalid.\n", optarg);
                    abort();
                }
                break;
            case 'q':
                params.dividendRate = atof(optarg);
                break;
            case 'l':
                params.jumpMean = atof(optarg);
                break;
            case '0':
                params.kouUpJumpProbability = atof(optarg);
                break;
            case '1':
                params.kouUpRate = atof(optarg);
                break;
            case '2':
                params.kouDownRate = atof(optarg);
                break;
            case 'y':
                params.mertonNormalStdev = atof(optarg);
                break;
            case 'S':
                params.startPrice = atof(optarg);
                break;
            case 'K':
                params.strikePrice = atof(optarg);
                break;
            case 'r':
                params.riskFreeRate = atof(optarg);
                break;
            case 'T':
                params.expiryTime = atof(optarg);
                break;
            case 'o':
                params.volatility = atof(optarg);
                break;
            case 'd':
                params.debug = true;
                break;
            case 'm':
                params.jumpType = Merton;
                break;
            case 'k':
                params.jumpType = Kou;
                break;
            case 'n':
                params.resolution = atoi(optarg);
                assert(isPowerOfTwo(params.resolution));
                break;
            case 't':
                params.timesteps = atoi(optarg);
                break;
            case 'v':
                params.verbose = true;
                break;
            case '?':
                break;
            default:
                abort();
        }
    }

    cudaCheck(params.debug);

    if (params.verbose) {
        printf("\nChecks finished. Starting option calculation...\n\n");
    }

    vector<double> assetPrices = assetPricesAtPayoff(params);
    vector<double> optionValues = optionValuesAtPayoff(params, assetPrices);

    if (params.verbose) {
        printf("\nComputing GPU results...\n");
    }
    computeGPU(params, assetPrices, optionValues);

    return EXIT_SUCCESS;
}

